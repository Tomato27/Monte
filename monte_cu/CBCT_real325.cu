#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#pragma comment(lib, "winmm.lib");

//とりあえず単一エネルギー

#define NUM 10000000 //発生光子数
#define ScatterNUM 5 //散乱回数

#define detector_x 65*5
#define detector_y 65*5

#define H 65*5
#define W 65*5
#define num_images 65
#define num_proj 360 

#define num_photon 10000

using namespace std; 

//energy125kv,al10mm

const char writeFileName0[] = "proj325_teth0cyu8e.raw";
const char writeFileName_m0[] = "map325_teth0cyu8e.raw";

const char writeFileName5[] = "proj325_teth5cyu8e.raw";
const char writeFileName_m5[] = "map325_teth5cyu8e.raw";

const char *H2O_c = "xcom2.txt";
const char *Ca_c = "Ca.txt";

const char geo_name[] = "spher01.raw";

char wname[11];

class csv
{
public:
  string fname;
  bool csv_get = false;
  csv(string filename, int sizex, int sizey);
};
csv::csv(string filename, int sizex, int sizey)
{
  fname = filename;
  //int sx=sizex;
  //int sy=sizey;
}

class photon
{
public:
  float x = 0, y = 0, z = 0;
  float x_p = 0, y_p = 0, z_p = 0;
  float before_vec0 = 0, before_vec1 = 0, before_vec2 = 0; //配列から変更
  float Energy;
  float theta = 0;
  float phi = 0;
  float length = 0;
  __device__ void delta_sampling(float mu_H2O, float mu_Ca, unsigned char *geometry, float sin_theta_a, float cos_theta_a, float sin_phi_a, float cos_phi_a, hiprandStateMRG32k3a *st);
};

//void readXcom(const char* csv_c, float* ab, float* coh, float* com, float* mu);
void readXcom();
void readxray();
void writeRawFile(const char fname[], const size_t size, const size_t num, void *image);
void writecsv(int height, int width, int *image, int *es, int num);

__global__ void projection(int per, float mu_H2O, float mu_Ca, unsigned char *geometry, int *image0, int *image5, hiprandStateMRG32k3a *state_gpu, float *ab_H2O, float *coh_H2O, float *com_H2O, float *mua_H2O,
                           float *ab_Ca, float *coh_Ca, float *com_Ca, float *mua_Ca, float *al10mm);
//後で散乱を足して，csv_H2Oを渡すように変える
__global__ void LaunchPhoton(hiprandStateMRG32k3a *state, int seed);
//__device__ void delta_sampling(photon p_gpu, float mu_H2O, float mu_Ca, unsigned char* geometry, float sin_theta_a, float cos_theta_a, float sin_phi_a, float cos_phi_a,hiprandStateMRG32k3a* state_gpu);
__global__ void RandStateGenerator(hiprandStateMRG32k3a *state_gpu);
void add_result(photon *p, float phi, int *image, int *es, int count, int ditector_index, int Energy, int a, int q);

float ab_H2O[201], coh_H2O[201], com_H2O[201], mua_H2O[201];
float ab_Ca[201], coh_Ca[201], com_Ca[201], mua_Ca[201];
float al2mm[251];
float al10mm[251];

int main(void)
{
  //cout<<"joxl";

  //int H=65,W=65,num_images=65 ,num_proj = 360;

  unsigned char *geometry = (unsigned char *)calloc(37 * 5 * 37 * 5 * num_images * 5, sizeof(unsigned char)); /** 原画像用配列 **/
  unsigned char *geometry_gpu;

  FILE *fpi; /** ファイルポインタ **/

  fpi = fopen(geo_name, "rb");
  fread(geometry, sizeof(unsigned char), 37 * 5 * 37 * 5 * num_images * 5, fpi); //phantomは0.1cm間隔

  hipMalloc((void **)&geometry_gpu, sizeof(unsigned char) * 37 * 5 * 37 * 5 * num_images * 5);
  hipMemcpy(geometry_gpu, geometry, sizeof(unsigned char) * 37 * 5 * 37 * 5 * num_images * 5, hipMemcpyHostToDevice);

  fclose(fpi);
  //int count_o9=0;

  int *image0 = (int *)calloc(detector_x * detector_y * 1 * num_proj, sizeof(int));     //処理用画像65*65
  int *image_out0 = (int *)calloc(detector_x * detector_y * 1 * num_proj, sizeof(int)); //処理用画像65*65

  int *image_gpu0;
  hipMalloc((void **)&image_gpu0, sizeof(int) * detector_y * detector_x * num_proj);
  hipMemcpy(image_gpu0, image0, sizeof(int) * detector_y * detector_x * num_proj, hipMemcpyHostToDevice);

  int *image5 = (int *)calloc(detector_x * detector_y * 1 * num_proj, sizeof(int));     //処理用画像65*65
  int *image_out5 = (int *)calloc(detector_x * detector_y * 1 * num_proj, sizeof(int)); //処理用画像65*65

  int *image_gpu5;
  hipMalloc((void **)&image_gpu5, sizeof(int) * detector_y * detector_x * num_proj);
  hipMemcpy(image_gpu5, image5, sizeof(int) * detector_y * detector_x * num_proj, hipMemcpyHostToDevice);

  //int i,
  int count = 0, sizex, sizey;
  int start_keV = 140, end_keV = 140;
  float sum_length = 0.0;
  float dens_H2O = 1.0; //水の密度i
  float dens_Ca = 1.550;

  /*線減衰係数の値をmuに代入*/
  sizex = 4; //線減衰係数配列のインデックス
  sizey = 200;

  readXcom();
  readxray();

  float *ab_H2O_gpu;
  float *coh_H2O_gpu;
  float *com_H2O_gpu;
  float *mua_H2O_gpu;

  float *ab_Ca_gpu;
  float *coh_Ca_gpu;
  float *com_Ca_gpu;
  float *mua_Ca_gpu;

  float *al2mm_gpu;
  float *al10mm_gpu;

  hipMalloc((void **)&ab_H2O_gpu, sizeof(float) * 201);
  hipMemcpy(ab_H2O_gpu, ab_H2O, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&coh_H2O_gpu, sizeof(float) * 201);
  hipMemcpy(coh_H2O_gpu, coh_H2O, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&com_H2O_gpu, sizeof(float) * 201);
  hipMemcpy(com_H2O_gpu, com_H2O, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&mua_H2O_gpu, sizeof(float) * 201);
  hipMemcpy(mua_H2O_gpu, mua_H2O, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&ab_Ca_gpu, sizeof(float) * 201);
  hipMemcpy(ab_Ca_gpu, ab_Ca, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&coh_Ca_gpu, sizeof(float) * 201);
  hipMemcpy(coh_Ca_gpu, coh_Ca, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&com_Ca_gpu, sizeof(float) * 201);
  hipMemcpy(com_Ca_gpu, com_Ca, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&mua_Ca_gpu, sizeof(float) * 201);
  hipMemcpy(mua_Ca_gpu, mua_Ca, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&al2mm_gpu, sizeof(float) * 251);
  hipMemcpy(al2mm_gpu, al2mm, sizeof(float) * 251, hipMemcpyHostToDevice);

  hipMalloc((void **)&al10mm_gpu, sizeof(float) * 251);
  hipMemcpy(al10mm_gpu, al10mm, sizeof(float) * 251, hipMemcpyHostToDevice);

  //int keV=140;
  //min_e=start_keV;

  int per = num_photon;

  int block_x_n = 1;
  int block_y_n = 1;

  //一番近い2のべき乗を求める
  while (true)
  {
    block_x_n *= 2;
    if (block_x_n >= detector_x)
    {
      break;
    }
  }

  while (true)
  {
    block_y_n *= 2;
    if (block_y_n >= detector_y)
    {
      break;
    }
  }
  cout << "\n blockx, block_y = (" << block_x_n << ", " << block_y_n << ")";

  // dim3変数の宣言
  dim3 blocks(block_x_n / 8, block_y_n / 8);
  dim3 threads(8, 8);


  for (int q = ScatterNUM; q <= ScatterNUM; q++)
  {
    for (int keV = start_keV; keV <= end_keV; keV++)
    {
      float Energy = start_keV;
      float mu_H2O = mua_H2O[(int)(Energy + 0.5)] * dens_H2O; //csv_H2O[3][(int)(Energy+0.5)]*dens_H2O;
      float mu_Ca = mua_Ca[(int)(Energy + 0.5)] * dens_Ca;    //csv_Ca[3][(int)(Energy+0.5)]*dens_Ca;
      float mu_Max = (mu_H2O <= mu_Ca) ? mu_Ca : mu_H2O;

      sum_length = 0;
      cout << "--------------------" << keV << "keV"
           << "--------------------\n";
      cout << "(mu_H2O, mu_Ca, mu_Max) = (" << mu_H2O << "," << mu_Ca << "," << mu_Max << ")" << endl;

      count = 0;

      //float theta=0., phi=0.;
      //float min_e=140.;
      //int primary_num=0;

      //int minus_num=0;
      //int coh_num=0;
      //int projection_step = 90;

      /*photon p;
      //p = new photon;
      photon p_gpu;
      //p_gpu = ;
      hipMalloc((void**)&p_gpu, sizeof(photon) * 1);
      hipMemcpy(p_gpu , &p ,sizeof(photon) * 1, hipMemcpyHostToDevice);*/
      //配列無い時は&が必要

      //シード値
      hiprandStateMRG32k3a *state_gpu;
      hipMalloc((void **)&state_gpu, sizeof(hiprandStateMRG32k3a) * detector_x * detector_y);

      RandStateGenerator<<<blocks, threads>>>(state_gpu);
      hipDeviceSynchronize();

      projection<<<blocks, threads>>>(per, mu_H2O, mu_Ca, geometry_gpu, image_gpu0, image_gpu5, state_gpu,
                                      ab_H2O_gpu, coh_H2O_gpu, com_H2O_gpu, mua_H2O_gpu, ab_Ca_gpu, coh_Ca_gpu, com_Ca_gpu, mua_Ca_gpu, al10mm_gpu);

      hipDeviceSynchronize();

      hipMemcpy(image_out0, image_gpu0, sizeof(int) * detector_x * detector_y * num_proj, hipMemcpyDeviceToHost);

      hipMemcpy(image_out5, image_gpu5, sizeof(int) * detector_x * detector_y * num_proj, hipMemcpyDeviceToHost);

      cout << "-----result-----\n";

      /*平均光路長の出力*/
      cout << "count = " << count << " / " << H * W * per << "\n\n";

      //sprintf(wname,"%s.raw",writeFileName);
      //cout<<"spok"<<endl;

      float *map_out0 = (float *)calloc(H * W * 1 * num_proj, sizeof(float)); //処理用画像65*65*360(real)
      float *map_out5 = (float *)calloc(H * W * 1 * num_proj, sizeof(float)); //処理用画像65*65*360(real)

      for (int a = 0; a < num_proj; a++)
      {
        for (int b = 0; b < H; b++)
        {
          for (int c = 0; c < W; c++)
          {
            if (image_out0[a * H * W + b * H + c] > per)
            { //||image_out[a*H*W + b*H + c]>2000){
              image_out0[a * H * W + b * H + c] = per;
            }
            if (image_out5[a * H * W + b * H + c] > per)
            { //||image_out[a*H*W + b*H + c]>2000){
              image_out5[a * H * W + b * H + c] = per;
            }
            if (image_out0[a * H * W + b * H + c] == 0)
            {
              image_out0[a * H * W + b * H + c] = 1;
            }
            if (image_out5[a * H * W + b * H + c] == 0)
            {
              image_out5[a * H * W + b * H + c] = 1;
            }
            //cout<<image_out[a*65*65 + b*65 + c]<<" "<<-log(image_out[a*65*65 + b*65 + c])+log(2000)<<endl;
            map_out0[a * H * W + b * H + c] = -log(image_out0[a * H * W + b * H + c]) + log(float(per));
            map_out5[a * H * W + b * H + c] = -log(image_out5[a * H * W + b * H + c]) + log(float(per));
          }
        }
      }

      writeRawFile(writeFileName0, sizeof(int), H * W * 1 * num_proj, image_out0);
      writeRawFile(writeFileName_m0, sizeof(float), H * W * 1 * num_proj, map_out0);

      writeRawFile(writeFileName5, sizeof(int), H * W * 1 * num_proj, image_out5);
      writeRawFile(writeFileName_m5, sizeof(float), H * W * 1 * num_proj, map_out5);
    }
  }
}

void readXcom()
{
  FILE *fp_h;
  fp_h = fopen(H2O_c, "r");

  if (fp_h == NULL)
  {
    printf("failed to open %s\n", H2O_c);
    exit(-1);
  }
  else
  {
    printf("%s \n", H2O_c);
    float buf[4];

    for (int i = 0; i <= 200; i++)
    {
      fscanf(fp_h, "%f\t%f\t%f\t%f", &coh_H2O[i], &com_H2O[i], &ab_H2O[i], &mua_H2O[i]);
      //printf("%f\t%f\t%f\t%f", coh_H2O[i], com_H2O[i], ab_H2O[i], mua_H2O[i]);
    }
  }
  fclose(fp_h);

  FILE *fp_c;
  int ret2 = 1;

  fp_c = fopen(Ca_c, "r");
  if (fp_c == NULL)
  {
    printf("failed to open %s\n", Ca_c);
    exit(-1);
  }
  printf("%s \n", Ca_c);

  for (int j = 0; j <= 200; j++)
  {
    //cout<<fp<<" ";
    fscanf(fp_c, "%f\t%f\t%f\t%f", &coh_Ca[j], &com_Ca[j], &ab_Ca[j], &mua_Ca[j]);
    //cout<<ab_Ca[j]<<" "<<coh_Ca[j]<<" "<<com_Ca[j]<<" "<<mua_Ca[j]<<endl;
  }

  fclose(fp_c);
}

void readxray()
{
  FILE *fp_h;
  int ret1 = 1;

  fp_h = fopen("125kv_al2mm.txt", "r");
  //printf("%d \n",fp_h[0]);
  if (fp_h == NULL)
  {
    printf("failed to open %s\n", "125kv_2mm.txt");
    exit(-1);
  }
  printf("125kv_al2mm.txt \n");

  for (int i = 0; i <= 250; i++)
  {
    fscanf(fp_h, "%f", &al2mm[i]);
    //printf("%f ",al2mm[i]);
  }

  fclose(fp_h);

  FILE *fp_c;
  int ret2 = 1;

  fp_c = fopen("125kv_al10mm.txt", "r");
  if (fp_c == NULL)
  {
    printf("failed to open 125kv_al10mm.txt\n");
    exit(-1);
  }
  printf("125kv_al10mm.txt ");

  for (int j = 0; j <= 250; j++)
  {
    fscanf(fp_h, "%f", &al10mm[j]);
    //printf("%f ",al10mm[j]);
  }

  fclose(fp_c);
}

void writeRawFile(const char fname[], const size_t size, const size_t num, void *image)
{
  // ファイルを開く
  FILE *fp = fopen(fname, "wb");

  // ファイルを開くことができなかった場合のエラー処理
  if (NULL == fp)
  {
    printf("failed to open %s\n", fname);
    exit(-1);
  }

  // データの書き出し
  //cout<<"loadhvoval";
  size_t ret = fwrite(image, size, num, fp);

  // データを書き込むことができなかった場合のエラー処理
  if (num != ret)
  {
    printf("failed to write %s\n", fname);
    fclose(fp);
    exit(-1);
  }

  // ファイルを閉じる
  fclose(fp);
}

__global__ void projection(int per, float mu_H2O, float mu_Ca, unsigned char *geometry, int *image0, int *image5, hiprandStateMRG32k3a *state_gpu,
                           float *ab_H2O, float *coh_H2O, float *com_H2O, float *mua_H2O,
                           float *ab_Ca, float *coh_Ca, float *com_Ca, float *mua_Ca, float *al10mm)
{
  //関数にditector_x, ditector_y渡す
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  int countp[325] = {0};

  float dens_H2O = 1.0; //水の密度i
  float dens_Ca = 1.550;

  int num_scatter = 0;

  if (j >= detector_x || i >= detector_y)
  {
    return;
  }
  int err = 0;

  int s_index = detector_y * i + j;

  hiprandStateMRG32k3a st = state_gpu[s_index];

  int projection_step = 1;
  int num_add = 0;

  const float detector_height = 16.25, Dso = 160., Dod = 60., Dsd = 220., pixel_size_d = 0.1;
  const float start_fantom = 10.1;

  for (int num_p = 0; num_p < num_proj; num_p += projection_step)
  { //投影数(xy平面で回転)for(int num_proj = 0; num_proj)
    for (int num_ph = 0; num_ph < per; num_ph++)
    {
      float cos_theta_a_new = 1., cos_phi_a_new = 0., sin_theta_a_new = 0., sin_phi_a_new = 1.;
      float cos_theta_a = 0., sin_theta_a = 1.;
      photon p1 = photon();
      //p1 = new photon;//p_gpu[0];

      p1.x = -Dso; //-16.25*sqrt(3)/2;
      p1.y = 0;
      p1.z = 0;

      //zlをマイナスに
      float xl = Dod,
            yl = detector_height - 0.5 * pixel_size_d /*ピクセル中心*/ - pixel_size_d * i,
            zl = detector_height - 0.5 * pixel_size_d - pixel_size_d * j;
            /*if(num_p == 0 && num_ph == 0 && j == 0){
              //atomicAdd(&countp[i],1);
              printf("%f, %f \n",yl,zl);
            }*/

      float cos_phi_a, sin_phi_a;
      float theta_a = 0.5 * M_PI - atan(zl / Dsd);

      float cos_theta_a2 = cos(theta_a), sin_theta_a2 = sin(theta_a);
      cos_theta_a = cos_theta_a2, sin_theta_a = sin_theta_a2;


      float Energy = 140; //start_keV
      float e_rand = hiprand_uniform(&st);
      for(int dec_e = 0; dec_e <250; dec_e++){
        if( al10mm[dec_e] <= e_rand  && e_rand <= al10mm[dec_e+1]){
          Energy = (dec_e+1)*0.5;
          break;
        }
      }
      //printf("Energy: %f", Energy);

      mu_H2O = mua_H2O[(int)(Energy + 0.5)] * dens_H2O;
      mu_Ca = mua_Ca[(int)(Energy + 0.5)] * dens_Ca;

      //printf("%f\n",mua_H2O[(int)(Energy + 0.5)]);
      //mu_H2O=csv_H2O[3][(int)(Energy+0.5)]*dens_H2O;-----------------------?
      //mu_Ca=csv_Ca[3][(int)(Energy+0.5)]*dens_Ca;

      float phia; // = (29.95-(59.9*j/64.))*M_PI/180.;
      //pixel中心から
      phia = atan(yl / Dsd);
      phia += M_PI * num_p / 180.;

      //int ditector_index = 180*phia/(2*M_PI);//検出器の検出画素のインデックス
      sin_phi_a = sin(phia);
      cos_phi_a = cos(phia);
      //cos_phi_a_new = cos_phi_a, sin_phi_a_new = sin_phi_a;

      //線源の回転
      float primary_x = p1.x * cos(M_PI * num_p / 180.) - p1.y * sin(M_PI * num_p / 180.);
      float primary_y = p1.x * sin(M_PI * num_p / 180.) + p1.y * cos(M_PI * num_p / 180.);

      p1.x = primary_x;
      p1.y = primary_y;

      //x=-start_fantomの位置まで飛ばす
      float photon_vec[3];                                                                      //光子の座標が入るベクトル
      photon_vec[0] = Dod * cos(M_PI * num_p / 180.) - yl * sin(M_PI * num_p / 180.) - primary_x; //60 - primary_x;//x
      photon_vec[1] = Dod * sin(M_PI * num_p / 180.) + yl * cos(M_PI * num_p / 180.) - primary_y; //yl - primary_y;//y
      photon_vec[2] = zl;

      //ここ書き換え必要
      float to = float(Dso - start_fantom) / Dsd;

      float to_ditector_x = p1.x + photon_vec[0];
      float to_ditector_y = p1.y + photon_vec[1];

      for (int to_n = 0; to_n < 3; to_n++){
        photon_vec[to_n] *= to;
      }

      p1.x += photon_vec[0];
      p1.y += photon_vec[1];
      p1.z += photon_vec[2];
      if(isnan(p1.x)||isnan(p1.y)||isnan(p1.z)){
        printf("%f,%f,%f\n",p1.x,p1.y,p1.z);
      }

      if(isnan(sin_theta_a2)||isnan(cos_theta_a2)||isnan(sin_phi_a)||isnan(cos_phi_a)){
        printf("%f,%f,%f, %f\n",sin_theta_a2, cos_theta_a2, sin_phi_a, cos_phi_a);
      }

      p1.delta_sampling(mu_H2O, mu_Ca, geometry, sin_theta_a2, cos_theta_a2, sin_phi_a, cos_phi_a, &st);
      //printf("%f\n\n",p1.x);
      //線源を元の座標に戻すために逆回転
      float tmp_x = p1.x;
      float tmp_y = p1.y;
      float tmp_z = p1.z;

      float x_r = tmp_x * cosf((-num_p * M_PI) / 180.) - tmp_y * sinf((-num_p * M_PI) / 180.);
      float y_r = tmp_x * sinf((-num_p * M_PI) / 180.) + tmp_y * cosf((-num_p * M_PI) / 180.);

      //p1.x =  tmp_x*cos(M_PI*-num_p/180) - tmp_y*sin(M_PI*-num_p/180);
      //p1.y =  tmp_x*sin(M_PI*-num_p/180) + tmp_y*cos(M_PI*-num_p/180);

      //if(num_ph==0&&num_p == 0){printf("%d %d %f %f %f %f %f\n", i, j , p1.x, p1.y, p1.z, to_ditector_y, to_ditector_x);}

      /*if(i==32&&j==32){
        printf(" %f %f ", x_r,y_r);
      }*/
      if (x_r >= Dod){ //1回目で検出器到着---------------primary-----------------&&yの範囲

        p1.x = x_r;
        p1.y = y_r;
        float d_z = ((p1.z - 0) / (p1.x - (-160))) * 60 + 160 * p1.z / (p1.x + 160);
        float d_y = ((p1.y - 0) / (p1.x - (-160))) * 60 + 160 * p1.y / (p1.x + 160);

        int result_y = -1 * (int(d_y * 10 - 325./2.)); //x,y変更
        int result_x = -1 * (int(d_z * 10 - 325./2.));
        if (result_y < 0 || result_y > 325 || result_x < 0 || result_x > 325)
        {
          err++;
        }

        //if(ScatterNUM == 0)
        //{
        num_add++;
        atomicAdd(&image0[num_p * detector_y * detector_x + result_y * detector_y + result_x], 1);
        atomicAdd(&image5[num_p * detector_y * detector_x + result_y * detector_y + result_x], 1);
        //if(i==32&&j==32)printf("%d add\n",num_add);
        //}

        //break;
      }
      else
      { //散乱
        num_scatter++;
        //if(i==32&&j==32)
        //printf("%d 散乱  ",num_ph);
        bool coh_flag = false;
        bool com_flag = false;

        for (int a = 0; a < ScatterNUM; a++)
        {
          //state_gpu[s_index] = st;///////////?
          float dens, ab, coh, com, mu;

          if (isnan(cos_theta_a) || isnan(cos_phi_a) || isnan(sin_theta_a) || isnan(sin_phi_a) ||
              isnan(cos_theta_a_new) || isnan(cos_phi_a_new) || isnan(sin_theta_a_new) || isnan(sin_phi_a_new))
          {
            printf("a %d %d %d \t",a, num_p,num_ph);
            printf("%f %f %f %f %f %f %f %f\n",cos_theta_a,cos_phi_a,sin_theta_a,sin_phi_a
            ,cos_theta_a_new, cos_phi_a_new, sin_theta_a_new, sin_phi_a_new);
            break;
          }

          float x_rotate_c = p1.x * cosf(-num_p * M_PI / 180) - p1.y * sinf(-num_p * M_PI / 180);
          float y_rotate_c = p1.x * sinf(-num_p * M_PI / 180) + p1.y * cosf(-num_p * M_PI / 180);
          if (x_rotate_c >= 60 || abs(y_rotate_c) >= 16.25 || abs(p1.z) >= 16.25)
          {
            //printf("break");
            break;
          }
          
          //if(pow((p1.y-5),2)+pow(p1.z,2) <= 9||pow((p1.y+3),2)+pow((p1.z+3),2) <= 9)

          if(pow((p1.y-5),2)+pow(p1.z,2) <= pow(1.5,2) || pow((p1.y+5),2)+pow(p1.z,2) <= pow(1.5,2)
	          || pow(p1.y,2)+pow((p1.z-5),2) <= pow(1.5,2) || pow(p1.y,2)+pow((p1.z+5),2) <= pow(1.5,2)
	          || pow((p1.y-5*sin(M_PI/4.)),2)+pow((p1.z-5*cos(M_PI/4.)),2) <= pow(1.5,2) || pow((p1.y+5*sin(M_PI/4.)),2)+pow((p1.z+5*cos(M_PI/4.)),2) <= pow(1.5,2)
	          || pow((p1.y-5*sin(M_PI*3./4.)),2)+pow((p1.z-5*cos(M_PI*3./4.)),2) <= pow(1.5,2) || pow((p1.y+5*sin(M_PI*3./4.)),2)+pow((p1.z+5*cos(M_PI*3./4.)),2) <= pow(1.5,2)
	        )
          {
            dens = dens_Ca;
            ab = ab_Ca[(int)(Energy+0.5)];
            coh = coh_Ca[(int)(Energy+0.5)];
            com = com_Ca[(int)(Energy+0.5)];
            mu =  mua_Ca[(int)(Energy+0.5)];
          }
          else
          {
            dens = dens_H2O;
            ab = ab_H2O[(int)(Energy + 0.5)];
            coh = coh_H2O[(int)(Energy + 0.5)];
            com = com_H2O[(int)(Energy + 0.5)];
            mu = mua_H2O[(int)(Energy + 0.5)];
          }

          float sc_rand = hiprand_uniform(&st); //////////?

          if (sc_rand <= ab / mu)
          { //光電効果
            //if(i==32&&j==32)printf("光電 ");
            break;
          }
          if (ab / mu < sc_rand && sc_rand <= (ab + coh) / mu)
          { //コヒーレント散乱----------------------------------------------------------------------------
            //if(i==32&&j==32)printf("coh ");
            //coh_num++;
            //num_scatter--;

            if (com_flag == false)
            {
              p1.delta_sampling(mu_H2O, mu_Ca, geometry, sin_theta_a2, cos_theta_a2, sin_phi_a, cos_phi_a, &st);
              coh_flag = true;
            }
            else
            { //過去にコンプトンあるなら，その時の角度を参照
              p1.delta_sampling(mu_H2O, mu_Ca, geometry, sin_theta_a_new, cos_theta_a_new, sin_phi_a_new, cos_phi_a_new, &st);
            }

            float x_rotate_ch = p1.x * cosf((-num_p * M_PI) / 180.) - p1.y * sinf((-num_p * M_PI) / 180.);
            float y_rotate_ch = p1.x * sinf((-num_p * M_PI) / 180.) + p1.y * cosf((-num_p * M_PI) / 180.);
            //printf("x_rotate_ch");

            float x_p_rotate_ch = p1.x_p * cosf((-num_p * M_PI) / 180.) - p1.y_p * sinf((-num_p * M_PI) / 180.);
            float y_p_rotate_ch = p1.x_p * sinf((-num_p * M_PI) / 180.) + p1.y_p * cosf((-num_p * M_PI) / 180.);
            //float x_p_rotate_ch = p1.x_p*cosf(M_PI*-num_p/180.) - p1.y_p*sin(M_PI*-num_p/180.);
            //float y_p_rotate_ch = p1.x_p*sin(M_PI*-num_p/180.) + p1.y_p*cos(M_PI*-num_p/180.);

            //float ditector_y=(10-x_p_rotate)*(y_rotate-y_p_rotate)/(x_rotate-x_p_rotate)+y_p_rotate;
            //float d_z_ch = ((p1.z - p1.z_p)/(x_rotate_ch-x_p_rotate_ch))*60 + (x_rotate_ch*p1.z_p - x_p_rotate_ch*p1.z)/(x_rotate_ch - x_p_rotate_ch);
            //float d_y_ch = ((y_rotate_ch - y_p_rotate_ch)/(x_rotate_ch-x_p_rotate_ch))*60 + (x_rotate_ch*y_p_rotate_ch - x_p_rotate_ch*y_rotate_ch)/(x_rotate_ch - x_p_rotate_ch);
            float d_z_ch = ((p1.z - p1.z_p) / (x_rotate_ch - x_p_rotate_ch)) * 60 + (x_rotate_ch * p1.z_p - x_p_rotate_ch * p1.z) / (x_rotate_ch - x_p_rotate_ch);
            float d_y_ch = ((y_rotate_ch - y_p_rotate_ch) / (x_rotate_ch - x_p_rotate_ch)) * 60 + (x_rotate_ch * y_p_rotate_ch - x_p_rotate_ch * y_rotate_ch) / (x_rotate_ch - x_p_rotate_ch);

            int result_x_ch = -1 * (int(d_z_ch * 10 - 325./2.)); //x,y変更
            int result_y_ch = -1 * (int(d_y_ch * 10 - 325./2.));

            if (x_rotate_ch >= 60 && result_y_ch <= 325 && result_x_ch <= 325)// && com_flag == true
            {
              atomicAdd(&image5[num_p * detector_y * detector_x + result_y_ch * detector_y + result_x_ch], 1);
              break;
            }
          }
          else
          { //コンプトン散乱
            //if(i==32&&j==32)printf("com ");//\n";

            //1:散乱角，エネルギー計算
            float lambda = 511.0 / Energy;

            float lambda_d = 0.;
            bool track_flag = true;

            while (track_flag)
            {
              //double r1=genrand_real3();
              float r1 = hiprand_uniform(&st);
              //r1=0.1;
              if (r1 < (lambda + 2.0) / (9.0 * lambda + 2.0))
              { //track1 <=or<?
                float r2 = hiprand_uniform(&st);
                //r2=0.2;
                float ro = 1.0 + (2.0 / lambda) * r2;
                float r3 = hiprand_uniform(&st);
                //r3=0.3;

                if (r3 <= 4.0 * ((1. / ro) - (1. / (ro * ro))))
                {
                  lambda_d = ro * lambda;
                  track_flag = false;
                }
              }
              else
              { //track2
                float r2 = hiprand_uniform(&st);
                float ro = (lambda + 2.) / (lambda + 2. * (1. - r2));
                float r3 = hiprand_uniform(&st);
                if (r3 <= 0.5 * (pow((lambda - ro * lambda + 1.), 2) + (1. / ro)))
                {
                  lambda_d = ro * lambda;
                  track_flag = false;
                }
              }
            }
            //lambda_d=lambda+0.1;

            float theta = acos(1. - (lambda_d - lambda)); ///----------------------何かまずいかも
            if (isnan(theta))
            {
              //printf("mz");
            }
            //float theta = atan(((lambda_d - lambda)*(lambda_d - lambda))/(lambda_d - lambda))
            float cos_theta = (1 - (lambda_d - lambda)); //cos(theta);//cos(0.5*M_PI + atan(50./220.));
            if (cos_theta < -1)
              cos_theta = -1;
            //0.5*_PIだとnanに
            float sin_theta = sqrt(1. - pow((cos_theta), 2));

            if (abs(cos_theta) > 1.0)
            {
              //cos_theta+=0.01;
              //printf(" %f %f\n", cos_theta,sin_theta);
            }

            Energy = 511. / lambda_d;
            //mu_H2O = csv_H2O[3][(int)(Energy+0.5)]*dens_H2O;
            //mu_Ca =  csv_Ca[3][(int)(Energy+0.5)]*dens_Ca;
            mu_H2O = mua_H2O[(int)(Energy + 0.5)] * dens_H2O;
            mu_Ca = mua_Ca[(int)(Energy + 0.5)] * dens_Ca;

            //2：方位角，光路長計算
            float phi = hiprand_uniform(&st) * 2. * M_PI;
            //乗算では.を忘れるな

            //3:相対座標→絶対座標
            if (com_flag == true)
            {
              sin_theta_a = sin_theta_a_new;
              cos_theta_a = cos_theta_a_new;
              cos_phi_a = cos_phi_a_new;
            }

            cos_theta_a_new = cos_theta_a * cos_theta - sin_theta_a * sin_theta * cos(phi); //cos(0.5*M_PI) -
            if (cos_theta_a_new < -1)cos_theta_a_new = -1;
            sin_theta_a_new = sqrt(1. - pow(cos_theta_a_new, 2)); //絶対座標系の新しい角度

            cos_phi_a_new = (cos_theta_a * cos_phi_a * sin_theta * cos(phi) + sin_theta_a * cos_phi_a * cos_theta - sin_phi_a * sin_theta * sin(phi)) / sin_theta_a_new;
            sin_phi_a_new = (cos_theta_a * sin_phi_a * sin_theta * cos(phi) + sin_theta_a * sin_phi_a * cos_theta + cos_phi_a * sin_theta * sin(phi)) / sin_theta_a_new;

            if (isnan(cos_phi_a_new) || isnan(sin_phi_a_new) || isnan(cos_theta_a_new) || isnan(sin_theta_a_new))
            {
              printf("%f, %f\n", cos_theta_a, sin_theta_a);
              printf("a: %d theta: %f, %f  phi; %f , %f\n", a, cos_theta_a_new, sin_theta_a_new, cos_phi_a_new, sin_phi_a_new);
              if (com_flag)
              {
                printf("com\n");
              }
              break;
            }

            com_flag = true;

            p1.delta_sampling(mu_H2O, mu_Ca, geometry, sin_theta_a_new, cos_theta_a_new, sin_phi_a_new, cos_phi_a_new, &st);
            //delta_sampling(p, mu_H2O, mu_Ca, geometry, sin_theta_a_new, cos_theta_a_new, sin_phi_a_new, cos_phi_a_new);

            p1.theta = theta;

            float v_length = sqrt(pow(p1.length * sin_theta_a_new * cos_phi_a_new, 2) + pow(p1.length * sin_theta_a_new * sin_phi_a_new, 2) + pow(p1.length * cos_theta_a_new, 2));
            p1.before_vec0 = p1.length * sin_theta_a_new * cos_phi_a_new / v_length;
            p1.before_vec1 = p1.length * sin_theta_a_new * sin_phi_a_new / v_length;
            p1.before_vec2 = p1.length * cos_theta_a_new / v_length;
            //vectorは長さ1に正規化

            //cout<<ditector_index<<" "<<phi_a_result<<endl;

            //float x_rotate=0, y_rotate=0, x_p_rotate=0, y_p_rotate=0;
            //座標の回転は今ついてる角度と逆方向に回さねば
            /*phi_a_result = num_p;//M_PI*210./180.;
              x_rotate = p1.x*cos(-phi_a_result) - p1.y*sin(-phi_a_result);
              y_rotate = p1.x*sin(-phi_a_result) + p1.y*cos(-phi_a_result);
              x_p_rotate = p1.x_p*cos(-phi_a_result) - p1.y_p*sin(-phi_a_result);
              y_p_rotate = p1.x_p*sin(-phi_a_result) + p1.y_p*cos(-phi_a_result);*/

            //cout<<p->x<<" "<<p->y<<" "<<p->z<<endl;

            //detector_y=(10-x_p_rotate)*(y_rotate-y_p_rotate)/(x_rotate-x_p_rotate)+y_p_rotate;//x==10の時のy座標

            //float d_z=(60-p1.x_p)*(p1.z-p1.z_p)/(p1.x-p1.x_p)+p1.z_p;
            //float d_y=(60-p1.x_p)*(p1.y-p1.y_p)/(p1.x-p1.x_p)+p1.y_p;

            float x_rotate = p1.x * cosf((-num_p * M_PI) / 180.) - p1.y * sinf((-num_p * M_PI) / 180.); //phi_a_resultから変更
            float y_rotate = p1.x * sinf((-num_p * M_PI) / 180.) + p1.y * cosf((-num_p * M_PI) / 180.);
            float x_p_rotate = p1.x_p * cosf((-num_p * M_PI) / 180.) - p1.y_p * sinf((-num_p * M_PI) / 180.);
            float y_p_rotate = p1.x_p * sinf((-num_p * M_PI) / 180.) + p1.y_p * cosf((-num_p * M_PI) / 180.);

            float d_z = ((p1.z - p1.z_p) / (x_rotate - x_p_rotate)) * 60 + (x_rotate * p1.z_p - x_p_rotate * p1.z) / (x_rotate - x_p_rotate);
            float d_y = ((y_rotate - y_p_rotate) / (x_rotate - x_p_rotate)) * 60 + (x_rotate * y_p_rotate - x_p_rotate * y_rotate) / (x_rotate - x_p_rotate);

            if (x_rotate >= 60 && abs(d_z) <= 16.25 && abs(d_y) <= 16.25)
            { //&&a==q-1){// && abs(p->z)<16.25 && abs(y_rotate)<16.25){// && x_rotate>10
              //検出器を通過したかcheck,今回は必ず検出器まで到達する

              int result_y = -1 * (int(d_y * 10 - 325./2.));
              int result_x = -1 * (int(d_z * 10- 325./2.));

              //count++;

              //image[+result_y*65+result_x]++;
              atomicAdd(&image5[num_p * detector_y * detector_x + result_y * detector_y + result_x], 1);
              break;
            }
          }
        }
      }
    }
  }
  state_gpu[s_index] = st;
  
  if (err != 0)
  {
    printf("err: %d", err);
  }
  //printf(" %d ",num_add);

  /*for(int out = 0; out<325;out++){
    printf("%f\n", countp[out]);
  }*/
}

__device__ void photon::delta_sampling(float mu_H2O, float mu_Ca, unsigned char *geometry, float sin_theta_a, float cos_theta_a, float sin_phi_a, float cos_phi_a, hiprandStateMRG32k3a *st)
{
  //printf("%f\n",x);
  float mu_max;
  mu_max = max(mu_H2O, mu_Ca); //2媒質の時
  //mu_max = mu_Ca; //1媒質の場合
  //printf("%f ",mu_max);
  bool loop_flag = true;
  bool air_flag = true;
  float x1 = x, y1 = y, z1 = z, length1 = length;
  float x2 = x, y2 = y, z2 = z, length3 = 0;
  x_p = x1, y_p = y1, z_p = z1;

  int num_itr = 0;
  //int geo_index;
  int check;
  if (isnan(sin_theta_a) || isnan(cos_theta_a) || isnan(sin_phi_a) || isnan(cos_phi_a))
  {
    printf("%f, %f , %f, %f\n", sin_theta_a, cos_theta_a, sin_phi_a, cos_theta_a);
    return;
  }

  while (loop_flag)
  {
    float beta = hiprand_uniform(st);
    float r = -log(beta) / mu_max;
    //printf("%f",r);

    x2 += r * sin_theta_a * cos_phi_a;
    y2 += r * sin_theta_a * sin_phi_a;
    z2 += r * cos_theta_a;
    length3 += r;
    //printf("%f, %f ,%f \n", x2,y2,z2);

    check = 0;

    //cout<<mu_Ca/mu_max;
    float nu = hiprand_uniform(st); //genrand_real3();

    /*if (x2 * x2 + y2 * y2 + z2 * z2 <= 100)//半径10cm球
    {
      check = 1;
    }*/
    /*if(x2 * x2 + y2 * y2 + z2 * z2 <= 100){//半径10球右Ca
      if(z2>=0){//Ca
        check = 2;
      }
      else{//h2o
        check = 2;
      }
    }*/
    /*if(y2 * y2 + z2 * z2 <= 100 && -10 <= x2 && x2 <= 10){//半径10円柱in 3cm,2cm Ca
      if(pow(y2-5,2)+pow(z2,2) <= 9||pow(y2+5,2)+pow(z2,2) <= 9
      || pow(y2,2)+pow(z2-5,2) <= 9||pow(y2,2)+pow(z2+5,2) <= 9){
        check = 2;
      }
      else{
        check = 1;
      }
    }*/
    /*if(x2 * x2 + y2 * y2 + z2 * z2 <= 16){//半径4球in 1.5cm Ca球
      if(pow((x2 - 2),2) + pow((y2),2) + pow(z2,2) <= 2.25){
        check = 2;
      }
      else{
        check = 1;
      }
    }*/
  if(y2 * y2 + z2 * z2 <= 100 && -10 <= x2 && x2 <= 10){//半径10円柱in 3cm Ca
      if(pow((y2-5),2)+pow(z2,2) <= pow(1.5,2) || pow((y2+5),2)+pow(z2,2) <= pow(1.5,2)
	|| pow(y2,2)+pow((z2-5),2) <= pow(1.5,2) || pow(y2,2)+pow((z2+5),2) <= pow(1.5,2)
	|| pow((y2-5*sin(M_PI/4.)),2)+pow((z2-5*cos(M_PI/4.)),2) <= pow(1.5,2) || pow((y2+5*sin(M_PI/4.)),2)+pow((z2+5*cos(M_PI/4.)),2) <= pow(1.5,2)
	|| pow((y2-5*sin(M_PI*3./4.)),2)+pow((z2-5*cos(M_PI*3./4.)),2) <= pow(1.5,2) || pow((y2+5*sin(M_PI*3./4.)),2)+pow((z2+5*cos(M_PI*3./4.)),2) <= pow(1.5,2)
	){

        check = 2;
      }
      else{
        check = 1;
      }
    }



    if (check == 0)
    { //空気の時
      //空気の時，直進し続ける
      if (abs(x2) >= 62 || abs(y2) >= 62 || abs(z2) >= 17)
      {
        loop_flag = false;
      }
      else
      {
        num_itr++;
      }
    }

    else if (check == 1)
    { //H2Oの時
      if (nu <= mu_H2O / mu_max)
      {
        loop_flag = false;
        num_itr++;
        //break;
      }
    }
//#if 1
    else{//Ca領域の時
      if(nu <=  mu_Ca/mu_max){
        loop_flag = false;
        //break;
      }
    }
//#endif

    if (num_itr > 100)
    {
      //printf("%d %f, %f, %f\n",num_itr, x2,y2,z2);
    }
  }

  x = x2;
  y = y2;
  z = z2;

  length = length3;
  //printf("%f\n",x);
}

__global__ void RandStateGenerator(hiprandStateMRG32k3a *state_gpu)
{
  int a = blockIdx.y * blockDim.y + threadIdx.y;
  int b = blockIdx.x * blockDim.x + threadIdx.x;


  if (a >= detector_x || b >= detector_y)
  {
    return;
  }
  int index = detector_x * a + b;

  hiprand_init(0, index, 0, &state_gpu[index]);
}

__global__ void LaunchPhoton(hiprandStateMRG32k3a *state, int seed)
{
  int a = blockIdx.y * blockDim.y + threadIdx.y;
  int b = blockIdx.x * blockDim.x + threadIdx.x;

  int index = detector_x * a + b;
  hiprand_init(seed, index, 0, &state[index]);
}

void add_result(photon *p, float phi, int *image, int *es, int count, int ditector_index, int Energy, int a, int q)
{
  float x_rotate = p->x * cos(-phi) - p->y * sin(-phi);
  float y_rotate = p->x * sin(-phi) + p->y * cos(-phi);
  float x_p_rotate = p->x_p * cos(-phi) - p->y_p * sin(-phi);
  float y_p_rotate = p->x_p * sin(-phi) + p->y_p * cos(-phi);

  float ditector_y = (10 - x_p_rotate) * (y_rotate - y_p_rotate) / (x_rotate - x_p_rotate) + y_p_rotate;

  if (x_rotate >= 16.25)
  {
    if (a != q - 1)
    { //特定の散乱回数の場合のみ検出
      return;
    }
    if (abs(ditector_y) > 10)
    {
      //cout<<ditector_index<<" "<<ditector_y<<endl;
    }
    image[ditector_index * 65 + (int)(ditector_y * 2 + 32.5)]++;
    count++;
    es[(int)Energy]++;
    return;
  }
}