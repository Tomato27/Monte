#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#pragma comment(lib, "winmm.lib");

#define num_photon 10000 //発生光子数
#define ScatterNUM 5 //散乱回数

#define detector_x 65*5
#define detector_y 65*5

#define H 65*5
#define W 65*5
#define num_images 65
#define num_proj 360 
#define diam 1.5*1.5

#define geoH 200 //ジオメトリ画像の高さ

using namespace std; 

const char writeFileName0[] = "proj325_teth0pmma8etim2.raw";
const char writeFileName_m0[] = "map325_teth0pmma8etim2.raw";

const char writeFileName5[] = "proj325_teth5pmma8etim2.raw";
const char writeFileName_m5[] = "map325_teth5pmma8etim2.raw";

const char *H2O_c = "xcom2.txt";
const char *Ca_c = "Ca.txt";
const char *PMMA_c = "PMMA.txt";

const char geo_name[] = "cyu8_2.raw";

char wname[11];

class csv
{
public:
  string fname;
  csv(string filename, int sizex, int sizey);
};
csv::csv(string filename, int sizex, int sizey)
{
  fname = filename;
}

class photon
{
public:
  float x = 0, y = 0, z = 0;
  float x_p = 0, y_p = 0, z_p = 0;
  float before_vec0 = 0, before_vec1 = 0, before_vec2 = 0; //配列から変更
  float Energy;
  float theta = 0;
  float phi = 0;
  float length = 0;
  __device__ void delta_sampling(float mu_H2O, float mu_Ca, float mu_PMMA ,unsigned char *geometry, float sin_theta_a, float cos_theta_a, float sin_phi_a, float cos_phi_a, hiprandStateMRG32k3a *st);
};


void readXcom();
void readxray();
void writeRawFile(const char fname[], const size_t size, const size_t num, void *image);
void writecsv(int height, int width, int *image, int *es, int num);

__global__ void projection(int per, float mu_H2O, float mu_Ca, float mu_PMMA ,unsigned char *geometry, int *image0, int *image5, hiprandStateMRG32k3a *state_gpu, 
                          float *ab_H2O, float *coh_H2O, float *com_H2O, float *mua_H2O,
                          float *ab_Ca, float *coh_Ca, float *com_Ca, float *mua_Ca, 
                          float *ab_PMMA, float *coh_PMMA, float *com_PMMA, float *mua_PMMA,
                          float *al10mm);

__global__ void LaunchPhoton(hiprandStateMRG32k3a *state, int seed);
__global__ void RandStateGenerator(hiprandStateMRG32k3a *state_gpu);
void add_result(photon *p, float phi, int *image, int *es, int count, int ditector_index, int Energy, int a, int q);

float ab_H2O[201], coh_H2O[201], com_H2O[201], mua_H2O[201];
float ab_Ca[201], coh_Ca[201], com_Ca[201], mua_Ca[201];
float ab_PMMA[201], coh_PMMA[201], com_PMMA[201], mua_PMMA[201];

float al2mm[251];
float al10mm[251];

int main(void)
{

  unsigned char *geometry = (unsigned char *)calloc(geoH * geoH * geoH, sizeof(unsigned char)); /** 原画像用配列 **/
  unsigned char *geometry_gpu;

  FILE *fpi; /** ファイルポインタ **/

  fpi = fopen(geo_name, "rb");
  fread(geometry, sizeof(unsigned char), geoH * geoH * geoH, fpi); //phantomは0.1cm間隔

  hipMalloc((void **)&geometry_gpu, sizeof(unsigned char) * geoH * geoH * geoH);
  hipMemcpy(geometry_gpu, geometry, sizeof(unsigned char) * geoH * geoH * geoH, hipMemcpyHostToDevice);

  fclose(fpi);

  int *image0 = (int *)calloc(detector_x * detector_y * 1 * num_proj, sizeof(int));     //処理用画像65*65
  int *image_out0 = (int *)calloc(detector_x * detector_y * 1 * num_proj, sizeof(int)); //処理用画像65*65

  int *image_gpu0;
  hipMalloc((void **)&image_gpu0, sizeof(int) * detector_y * detector_x * num_proj);
  hipMemcpy(image_gpu0, image0, sizeof(int) * detector_y * detector_x * num_proj, hipMemcpyHostToDevice);

  int *image5 = (int *)calloc(detector_x * detector_y * 1 * num_proj, sizeof(int));     //処理用画像65*65
  int *image_out5 = (int *)calloc(detector_x * detector_y * 1 * num_proj, sizeof(int)); //処理用画像65*65

  int *image_gpu5;
  hipMalloc((void **)&image_gpu5, sizeof(int) * detector_y * detector_x * num_proj);
  hipMemcpy(image_gpu5, image5, sizeof(int) * detector_y * detector_x * num_proj, hipMemcpyHostToDevice);

  int count = 0;
  int start_keV = 140, end_keV = 140;
  float dens_H2O = 1.0; //水の密度i
  float dens_Ca = 1.550;
  float dens_PMMA = 1.18;

  readXcom();
  readxray();

  float *ab_H2O_gpu;
  float *coh_H2O_gpu;
  float *com_H2O_gpu;
  float *mua_H2O_gpu;

  float *ab_Ca_gpu;
  float *coh_Ca_gpu;
  float *com_Ca_gpu;
  float *mua_Ca_gpu;

  float *ab_PMMA_gpu;
  float *coh_PMMA_gpu;
  float *com_PMMA_gpu;
  float *mua_PMMA_gpu;

  float *al2mm_gpu;
  float *al10mm_gpu;

  hipMalloc((void **)&ab_H2O_gpu, sizeof(float) * 201);
  hipMemcpy(ab_H2O_gpu, ab_H2O, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&coh_H2O_gpu, sizeof(float) * 201);
  hipMemcpy(coh_H2O_gpu, coh_H2O, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&com_H2O_gpu, sizeof(float) * 201);
  hipMemcpy(com_H2O_gpu, com_H2O, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&mua_H2O_gpu, sizeof(float) * 201);
  hipMemcpy(mua_H2O_gpu, mua_H2O, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&ab_Ca_gpu, sizeof(float) * 201);
  hipMemcpy(ab_Ca_gpu, ab_Ca, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&coh_Ca_gpu, sizeof(float) * 201);
  hipMemcpy(coh_Ca_gpu, coh_Ca, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&com_Ca_gpu, sizeof(float) * 201);
  hipMemcpy(com_Ca_gpu, com_Ca, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&mua_Ca_gpu, sizeof(float) * 201);
  hipMemcpy(mua_Ca_gpu, mua_Ca, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&ab_PMMA_gpu, sizeof(float) * 201);
  hipMemcpy(ab_PMMA_gpu, ab_PMMA, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&coh_PMMA_gpu, sizeof(float) * 201);
  hipMemcpy(coh_PMMA_gpu, coh_PMMA, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&com_PMMA_gpu, sizeof(float) * 201);
  hipMemcpy(com_PMMA_gpu, com_PMMA, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&mua_PMMA_gpu, sizeof(float) * 201);
  hipMemcpy(mua_PMMA_gpu, mua_PMMA, sizeof(float) * 201, hipMemcpyHostToDevice);

  hipMalloc((void **)&al2mm_gpu, sizeof(float) * 251);
  hipMemcpy(al2mm_gpu, al2mm, sizeof(float) * 251, hipMemcpyHostToDevice);

  hipMalloc((void **)&al10mm_gpu, sizeof(float) * 251);
  hipMemcpy(al10mm_gpu, al10mm, sizeof(float) * 251, hipMemcpyHostToDevice);

  int per = num_photon;

  int block_x_n = 1;
  int block_y_n = 1;

  //一番近い2のべき乗を求める
  while (true)
  {
    block_x_n *= 2;
    if (block_x_n >= detector_x)
    {
      break;
    }
  }

  while (true)
  {
    block_y_n *= 2;
    if (block_y_n >= detector_y)
    {
      break;
    }
  }
  cout << "\n blockx, block_y = (" << block_x_n << ", " << block_y_n << ")";

  // dim3変数の宣言
  dim3 blocks(block_x_n / 8, block_y_n / 8);
  dim3 threads(8, 8);


  for (int q = ScatterNUM; q <= ScatterNUM; q++)
  {
    for (int keV = start_keV; keV <= end_keV; keV++)
    {
      float Energy = start_keV;
      float mu_H2O = mua_H2O[(int)(Energy + 0.5)] * dens_H2O; //csv_H2O[3][(int)(Energy+0.5)]*dens_H2O;
      float mu_Ca = mua_Ca[(int)(Energy + 0.5)] * dens_Ca;    //csv_Ca[3][(int)(Energy+0.5)]*dens_Ca;
      float mu_PMMA = mua_PMMA[(int)(Energy + 0.5)] * dens_PMMA;
      
      float mu_Max = (mu_H2O <= mu_Ca) ? mu_Ca : mu_H2O;
      mu_Max = (mu_Max <= mu_PMMA) ? mu_PMMA : mu_Max;

      //sum_length = 0;
      cout << "--------------------" << keV << "keV"
           << "--------------------\n";
      cout << "(mu_H2O, mu_Ca, mu_PMMA ,mu_Max) = (" << mu_H2O << "," << mu_Ca << "," << mu_PMMA << " " << mu_Max << ")" << endl;

      count = 0;

      //シード値
      hiprandStateMRG32k3a *state_gpu;
      hipMalloc((void **)&state_gpu, sizeof(hiprandStateMRG32k3a) * detector_x * detector_y);

      RandStateGenerator<<<blocks, threads>>>(state_gpu);
      hipDeviceSynchronize();

      projection<<<blocks, threads>>>(per, mu_H2O, mu_Ca, mu_PMMA ,geometry_gpu, image_gpu0, image_gpu5, state_gpu,
                                      ab_H2O_gpu, coh_H2O_gpu, com_H2O_gpu, mua_H2O_gpu, 
                                      ab_Ca_gpu, coh_Ca_gpu, com_Ca_gpu, mua_Ca_gpu, 
                                      ab_PMMA_gpu, coh_PMMA_gpu, com_PMMA_gpu, mua_PMMA_gpu,
                                      al10mm_gpu);

      hipDeviceSynchronize();

      hipMemcpy(image_out0, image_gpu0, sizeof(int) * detector_x * detector_y * num_proj, hipMemcpyDeviceToHost);

      hipMemcpy(image_out5, image_gpu5, sizeof(int) * detector_x * detector_y * num_proj, hipMemcpyDeviceToHost);

      cout << "-----result-----\n";

      /*平均光路長の出力*/
      cout << "count = " << count << " / " << H * W * per << "\n\n";

      //sprintf(wname,"%s.raw",writeFileName);
      //cout<<"spok"<<endl;

      float *map_out0 = (float *)calloc(H * W * 1 * num_proj, sizeof(float)); //処理用画像65*65*360(real)
      float *map_out5 = (float *)calloc(H * W * 1 * num_proj, sizeof(float)); //処理用画像65*65*360(real)

      for (int a = 0; a < num_proj; a++)
      {
        for (int b = 0; b < H; b++)
        {
          for (int c = 0; c < W; c++)
          {
            if (image_out0[a * H * W + b * H + c] > per)
            { //||image_out[a*H*W + b*H + c]>2000){
              image_out0[a * H * W + b * H + c] = per;
            }
            if (image_out5[a * H * W + b * H + c] > per)
            { //||image_out[a*H*W + b*H + c]>2000){
              image_out5[a * H * W + b * H + c] = per;
            }
            if (image_out0[a * H * W + b * H + c] == 0)
            {
              image_out0[a * H * W + b * H + c] = 1;
            }
            if (image_out5[a * H * W + b * H + c] == 0)
            {
              image_out5[a * H * W + b * H + c] = 1;
            }
            //cout<<image_out[a*65*65 + b*65 + c]<<" "<<-log(image_out[a*65*65 + b*65 + c])+log(2000)<<endl;
            map_out0[a * H * W + b * H + c] = -log(image_out0[a * H * W + b * H + c]) + log(float(per));
            map_out5[a * H * W + b * H + c] = -log(image_out5[a * H * W + b * H + c]) + log(float(per));
          }
        }
      }

      writeRawFile(writeFileName0, sizeof(int), H * W * 1 * num_proj, image_out0);
      writeRawFile(writeFileName_m0, sizeof(float), H * W * 1 * num_proj, map_out0);

      writeRawFile(writeFileName5, sizeof(int), H * W * 1 * num_proj, image_out5);
      writeRawFile(writeFileName_m5, sizeof(float), H * W * 1 * num_proj, map_out5);
    }
  }
}

void readXcom()
{
  FILE *fp_h;
  fp_h = fopen(H2O_c, "r");

  if (fp_h == NULL)
  {
    printf("failed to open %s\n", H2O_c);
    exit(-1);
  }
  else
  {
    printf("%s \n", H2O_c);
    for (int i = 0; i <= 200; i++)
    {
      fscanf(fp_h, "%f\t%f\t%f\t%f", &coh_H2O[i], &com_H2O[i], &ab_H2O[i], &mua_H2O[i]);
    }
  }
  fclose(fp_h);

  FILE *fp_c;

  fp_c = fopen(Ca_c, "r");
  if (fp_c == NULL)
  {
    printf("failed to open %s\n", Ca_c);
    exit(-1);
  }
  printf("%s \n", Ca_c);

  for (int j = 0; j <= 200; j++)
  {
    fscanf(fp_c, "%f\t%f\t%f\t%f", &coh_Ca[j], &com_Ca[j], &ab_Ca[j], &mua_Ca[j]);
  }

  fclose(fp_c);

  FILE *fp_p;
  fp_p = fopen(PMMA_c, "r");

  if (fp_p == NULL)
  {
    printf("failed to open %s\n", PMMA_c);
    exit(-1);
  }
  else
  {
    printf("%s \n", PMMA_c);

    for (int i = 0; i <= 200; i++)
    {
      fscanf(fp_p, "%f\t%f\t%f\t%f", &coh_PMMA[i], &com_PMMA[i], &ab_PMMA[i], &mua_PMMA[i]);
      //printf("%f\t%f\t%f\t%f", coh_H2O[i], com_H2O[i], ab_H2O[i], mua_H2O[i]);
    }
  }
  fclose(fp_h);
}

void readxray()
{
  FILE *fp_h;

  fp_h = fopen("125kv_al2mm.txt", "r");
  //printf("%d \n",fp_h[0]);
  if (fp_h == NULL)
  {
    printf("failed to open %s\n", "125kv_2mm.txt");
    exit(-1);
  }
  printf("125kv_al2mm.txt \n");

  for (int i = 0; i <= 250; i++)
  {
    fscanf(fp_h, "%f", &al2mm[i]);
    //printf("%f ",al2mm[i]);
  }

  fclose(fp_h);

  FILE *fp_c;

  fp_c = fopen("125kv_al10mm.txt", "r");
  if (fp_c == NULL)
  {
    printf("failed to open 125kv_al10mm.txt\n");
    exit(-1);
  }
  printf("125kv_al10mm.txt ");

  for (int j = 0; j <= 250; j++)
  {
    fscanf(fp_h, "%f", &al10mm[j]);
    //printf("%f ",al10mm[j]);
  }

  fclose(fp_c);
}

void writeRawFile(const char fname[], const size_t size, const size_t num, void *image)
{
  // ファイルを開く
  FILE *fp = fopen(fname, "wb");

  // ファイルを開くことができなかった場合のエラー処理
  if (NULL == fp)
  {
    printf("failed to open %s\n", fname);
    exit(-1);
  }

  // データの書き出し
  //cout<<"loadhvoval";
  size_t ret = fwrite(image, size, num, fp);

  // データを書き込むことができなかった場合のエラー処理
  if (num != ret)
  {
    printf("failed to write %s\n", fname);
    fclose(fp);
    exit(-1);
  }

  // ファイルを閉じる
  fclose(fp);
}

__global__ void projection(int per, float mu_H2O, float mu_Ca, float mu_PMMA,unsigned char *geometry, int *image0, int *image5, hiprandStateMRG32k3a *state_gpu,
                           float *ab_H2O, float *coh_H2O, float *com_H2O, float *mua_H2O,
                           float *ab_Ca, float *coh_Ca, float *com_Ca, float *mua_Ca, 
                           float *ab_PMMA, float *coh_PMMA, float *com_PMMA, float *mua_PMMA, 
                           float *al10mm)
{
  //関数にditector_x, ditector_y渡す
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  //int countp[325] = {0};

  float dens_H2O = 1.0; //水の密度i
  float dens_Ca = 1.550;
  float dens_PMMA = 1.18;

  int num_scatter = 0;
  /*for(int w=0;w<200*200;w++){
    if(i==0&&j==0)printf("%d", int(geometry[w]));
  }*/

  if (j >= detector_x || i >= detector_y)
  {
    return;
  }
  int err = 0;

  int s_index = detector_y * i + j;

  hiprandStateMRG32k3a st = state_gpu[s_index];

  int projection_step = 1;
  int num_add = 0;

  const float detector_height = 16.25, Dso = 160., Dod = 60., Dsd = 220., pixel_size_d = 0.1;
  const float start_fantom = 10.1;

  for (int num_p = 0; num_p < num_proj; num_p += projection_step)
  { //投影数(xy平面で回転)for(int num_proj = 0; num_proj)
    for (int num_ph = 0; num_ph < per; num_ph++)
    {
      float cos_theta_a_new = 1., cos_phi_a_new = 0., sin_theta_a_new = 0., sin_phi_a_new = 1.;
      float cos_theta_a = 0., sin_theta_a = 1.;
      photon p1 = photon();
      //p1 = new photon;//p_gpu[0];

      p1.x = -Dso; //-16.25*sqrt(3)/2;
      p1.y = 0;
      p1.z = 0;

      //zlをマイナスに
      float yl = detector_height - 0.5 * pixel_size_d /*ピクセル中心*/ - pixel_size_d * i,
            zl = detector_height - 0.5 * pixel_size_d - pixel_size_d * j;
            //xl = Dod;
            /*if(num_p == 0 && num_ph == 0 && j == 0){
              //atomicAdd(&countp[i],1);
              printf("%f, %f \n",yl,zl);
            }*/

      float cos_phi_a, sin_phi_a;
      float theta_a = 0.5 * M_PI - atan(zl / Dsd);

      float cos_theta_a2 = cos(theta_a), sin_theta_a2 = sin(theta_a);
      cos_theta_a = cos_theta_a2, sin_theta_a = sin_theta_a2;


      float Energy = 140; //start_keV
      float e_rand = hiprand_uniform(&st);
      for(int dec_e = 0; dec_e <250; dec_e++){
        if( al10mm[dec_e] <= e_rand  && e_rand <= al10mm[dec_e+1]){
          Energy = (dec_e+1)*0.5;
          break;
        }
      }
      //printf("Energy: %f", Energy);

      mu_H2O = mua_H2O[(int)(Energy + 0.5)] * dens_H2O;
      mu_Ca = mua_Ca[(int)(Energy + 0.5)] * dens_Ca;
      mu_PMMA = mua_PMMA[(int)(Energy + 0.5)] * dens_PMMA;

      float phia; // = (29.95-(59.9*j/64.))*M_PI/180.;
      //pixel中心から
      phia = atan(yl / Dsd);
      phia += M_PI * num_p / 180.;

      //int ditector_index = 180*phia/(2*M_PI);//検出器の検出画素のインデックス
      sin_phi_a = sin(phia);
      cos_phi_a = cos(phia);
      //cos_phi_a_new = cos_phi_a, sin_phi_a_new = sin_phi_a;

      //線源の回転
      float primary_x = p1.x * cos(M_PI * num_p / 180.) - p1.y * sin(M_PI * num_p / 180.);
      float primary_y = p1.x * sin(M_PI * num_p / 180.) + p1.y * cos(M_PI * num_p / 180.);

      p1.x = primary_x;
      p1.y = primary_y;

      //x=-start_fantomの位置まで飛ばす
      float photon_vec[3];                                                                      //光子の座標が入るベクトル
      photon_vec[0] = Dod * cos(M_PI * num_p / 180.) - yl * sin(M_PI * num_p / 180.) - primary_x; //60 - primary_x;//x
      photon_vec[1] = Dod * sin(M_PI * num_p / 180.) + yl * cos(M_PI * num_p / 180.) - primary_y; //yl - primary_y;//y
      photon_vec[2] = zl;

      //ここ書き換え必要
      float to = float(Dso - start_fantom) / Dsd;

      //float to_ditector_x = p1.x + photon_vec[0];
      //float to_ditector_y = p1.y + photon_vec[1];

      for (int to_n = 0; to_n < 3; to_n++){
        photon_vec[to_n] *= to;
      }

      p1.x += photon_vec[0];
      p1.y += photon_vec[1];
      p1.z += photon_vec[2];
      if(isnan(p1.x)||isnan(p1.y)||isnan(p1.z)){
        printf("%f,%f,%f\n",p1.x,p1.y,p1.z);
      }

      if(isnan(sin_theta_a2)||isnan(cos_theta_a2)||isnan(sin_phi_a)||isnan(cos_phi_a)){
        printf("%f,%f,%f, %f\n",sin_theta_a2, cos_theta_a2, sin_phi_a, cos_phi_a);
      }

      p1.delta_sampling(mu_H2O, mu_Ca, mu_PMMA, geometry, sin_theta_a2, cos_theta_a2, sin_phi_a, cos_phi_a, &st);
      //printf("%f\n\n",p1.x);
      //線源を元の座標に戻すために逆回転
      float tmp_x = p1.x;
      float tmp_y = p1.y;
      //float tmp_z = p1.z;

      float x_r = tmp_x * cosf((-num_p * M_PI) / 180.) - tmp_y * sinf((-num_p * M_PI) / 180.);
      float y_r = tmp_x * sinf((-num_p * M_PI) / 180.) + tmp_y * cosf((-num_p * M_PI) / 180.);

      //p1.x =  tmp_x*cos(M_PI*-num_p/180) - tmp_y*sin(M_PI*-num_p/180);
      //p1.y =  tmp_x*sin(M_PI*-num_p/180) + tmp_y*cos(M_PI*-num_p/180);

      //if(num_ph==0&&num_p == 0){printf("%d %d %f %f %f %f %f\n", i, j , p1.x, p1.y, p1.z, to_ditector_y, to_ditector_x);}

      /*if(i==32&&j==32){
        printf(" %f %f ", x_r,y_r);
      }*/
      if (x_r >= Dod){ //1回目で検出器到着---------------primary-----------------&&yの範囲

        p1.x = x_r;
        p1.y = y_r;
        float d_z = ((p1.z - 0) / (p1.x - (-160))) * 60 + 160 * p1.z / (p1.x + 160);
        float d_y = ((p1.y - 0) / (p1.x - (-160))) * 60 + 160 * p1.y / (p1.x + 160);

        int result_y = -1 * (int(d_y * 10 - 325./2.)); //x,y変更
        int result_x = -1 * (int(d_z * 10 - 325./2.));
        if (result_y < 0 || result_y > 325 || result_x < 0 || result_x > 325)
        {
          err++;
        }

        //if(ScatterNUM == 0)
        //{
        num_add++;
        atomicAdd(&image0[num_p * detector_y * detector_x + result_y * detector_y + result_x], 1);
        atomicAdd(&image5[num_p * detector_y * detector_x + result_y * detector_y + result_x], 1);
        //if(i==32&&j==32)printf("%d add\n",num_add);
        //}

        //break;
      }
      else
      { //散乱
        num_scatter++;
        //if(i==32&&j==32)
        //printf("%d 散乱  ",num_ph);
        bool coh_flag = false;
        bool com_flag = false;

        for (int a = 0; a < ScatterNUM; a++)
        {
          //state_gpu[s_index] = st;///////////?
          float dens, ab, coh, com, mu;

          if (isnan(cos_theta_a) || isnan(cos_phi_a) || isnan(sin_theta_a) || isnan(sin_phi_a) ||
              isnan(cos_theta_a_new) || isnan(cos_phi_a_new) || isnan(sin_theta_a_new) || isnan(sin_phi_a_new))
          {
            printf("a %d %d %d \t",a, num_p,num_ph);
            printf("%f %f %f %f %f %f %f %f\n",cos_theta_a,cos_phi_a,sin_theta_a,sin_phi_a
            ,cos_theta_a_new, cos_phi_a_new, sin_theta_a_new, sin_phi_a_new);
            break;
          }

          float x_rotate_c = p1.x * cosf(-num_p * M_PI / 180) - p1.y * sinf(-num_p * M_PI / 180);
          float y_rotate_c = p1.x * sinf(-num_p * M_PI / 180) + p1.y * cosf(-num_p * M_PI / 180);
          if (x_rotate_c >= 60 || abs(y_rotate_c) >= 16.25 || abs(p1.z) >= 16.25)
          {
            //printf("break");
            break;
          }

    //条件が多いものを上に書くことで上でif文が切れる確率が上がり，計算量短縮につながりそう
    //pow(hoge,2)からべた書きに変更．数億回計算する事を考えると気になるので変更

    if(int(geometry[int((p1.z+10)*10)*geoH*geoH + int((p1.y+10)*10)*geoH + int((p1.x+10)*10)]) == 1)
    {
        dens = dens_H2O;
        ab = ab_H2O[(int)(Energy + 0.5)];
        coh = coh_H2O[(int)(Energy + 0.5)];
        com = com_H2O[(int)(Energy + 0.5)];
        mu = mua_H2O[(int)(Energy + 0.5)];
    }
    else if(int(geometry[int((p1.z+10)*10)*geoH*geoH + int((p1.y+10)*10)*geoH + int((p1.x+10)*10)]) == 2)
    {
        dens = dens_Ca;
        ab = ab_Ca[(int)(Energy+0.5)];
        coh = coh_Ca[(int)(Energy+0.5)];
        com = com_Ca[(int)(Energy+0.5)];
        mu =  mua_Ca[(int)(Energy+0.5)];   
    }
    else{
          dens = dens_PMMA;
          ab = ab_PMMA[(int)(Energy+0.5)];
          coh = coh_PMMA[(int)(Energy+0.5)];
          com = com_PMMA[(int)(Energy+0.5)];
          mu =  mua_PMMA[(int)(Energy+0.5)];
    }


          float sc_rand = hiprand_uniform(&st); //////////?

          if (sc_rand <= ab / mu)
          { //光電効果
            //if(i==32&&j==32)printf("光電 ");
            break;
          }
          if (ab / mu < sc_rand && sc_rand <= (ab + coh) / mu)
          { //コヒーレント散乱----------------------------------------------------------------------------
            //if(i==32&&j==32)printf("coh ");
            //coh_num++;
            //num_scatter--;

            if (com_flag == false)
            {
              p1.delta_sampling(mu_H2O, mu_Ca, mu_PMMA, geometry, sin_theta_a2, cos_theta_a2, sin_phi_a, cos_phi_a, &st);
              coh_flag = true;
            }
            else
            { //過去にコンプトンあるなら，その時の角度を参照
              p1.delta_sampling(mu_H2O, mu_Ca, mu_PMMA, geometry, sin_theta_a_new, cos_theta_a_new, sin_phi_a_new, cos_phi_a_new, &st);
            }

            float x_rotate_ch = p1.x * cosf((-num_p * M_PI) / 180.) - p1.y * sinf((-num_p * M_PI) / 180.);
            float y_rotate_ch = p1.x * sinf((-num_p * M_PI) / 180.) + p1.y * cosf((-num_p * M_PI) / 180.);
            //printf("x_rotate_ch");

            float x_p_rotate_ch = p1.x_p * cosf((-num_p * M_PI) / 180.) - p1.y_p * sinf((-num_p * M_PI) / 180.);
            float y_p_rotate_ch = p1.x_p * sinf((-num_p * M_PI) / 180.) + p1.y_p * cosf((-num_p * M_PI) / 180.);
            //float x_p_rotate_ch = p1.x_p*cosf(M_PI*-num_p/180.) - p1.y_p*sin(M_PI*-num_p/180.);
            //float y_p_rotate_ch = p1.x_p*sin(M_PI*-num_p/180.) + p1.y_p*cos(M_PI*-num_p/180.);

            //float ditector_y=(10-x_p_rotate)*(y_rotate-y_p_rotate)/(x_rotate-x_p_rotate)+y_p_rotate;
            //float d_z_ch = ((p1.z - p1.z_p)/(x_rotate_ch-x_p_rotate_ch))*60 + (x_rotate_ch*p1.z_p - x_p_rotate_ch*p1.z)/(x_rotate_ch - x_p_rotate_ch);
            //float d_y_ch = ((y_rotate_ch - y_p_rotate_ch)/(x_rotate_ch-x_p_rotate_ch))*60 + (x_rotate_ch*y_p_rotate_ch - x_p_rotate_ch*y_rotate_ch)/(x_rotate_ch - x_p_rotate_ch);
            float d_z_ch = ((p1.z - p1.z_p) / (x_rotate_ch - x_p_rotate_ch)) * 60 + (x_rotate_ch * p1.z_p - x_p_rotate_ch * p1.z) / (x_rotate_ch - x_p_rotate_ch);
            float d_y_ch = ((y_rotate_ch - y_p_rotate_ch) / (x_rotate_ch - x_p_rotate_ch)) * 60 + (x_rotate_ch * y_p_rotate_ch - x_p_rotate_ch * y_rotate_ch) / (x_rotate_ch - x_p_rotate_ch);

            int result_x_ch = -1 * (int(d_z_ch * 10 - 325./2.)); //x,y変更
            int result_y_ch = -1 * (int(d_y_ch * 10 - 325./2.));

            if (x_rotate_ch >= 60 && result_y_ch <= 325 && result_x_ch <= 325)// && com_flag == true
            {
              atomicAdd(&image5[num_p * detector_y * detector_x + result_y_ch * detector_y + result_x_ch], 1);
              break;
            }
          }
          else
          { //コンプトン散乱
            //if(i==32&&j==32)printf("com ");//\n";

            //1:散乱角，エネルギー計算
            float lambda = 511.0 / Energy;

            float lambda_d = 0.;
            bool track_flag = true;

            while (track_flag)
            {
              //double r1=genrand_real3();
              float r1 = hiprand_uniform(&st);
              //r1=0.1;
              if (r1 < (lambda + 2.0) / (9.0 * lambda + 2.0))
              { //track1 <=or<?
                float r2 = hiprand_uniform(&st);
                //r2=0.2;
                float ro = 1.0 + (2.0 / lambda) * r2;
                float r3 = hiprand_uniform(&st);
                //r3=0.3;

                if (r3 <= 4.0 * ((1. / ro) - (1. / (ro * ro))))
                {
                  lambda_d = ro * lambda;
                  track_flag = false;
                }
              }
              else
              { //track2
                float r2 = hiprand_uniform(&st);
                float ro = (lambda + 2.) / (lambda + 2. * (1. - r2));
                float r3 = hiprand_uniform(&st);
                if (r3 <= 0.5 * (pow((lambda - ro * lambda + 1.), 2) + (1. / ro)))
                {
                  lambda_d = ro * lambda;
                  track_flag = false;
                }
              }
            }
            //lambda_d=lambda+0.1;

            float theta = acos(1. - (lambda_d - lambda)); ///----------------------何かまずいかも
            if (isnan(theta))
            {
              //printf("mz");
            }
            //float theta = atan(((lambda_d - lambda)*(lambda_d - lambda))/(lambda_d - lambda))
            float cos_theta = (1 - (lambda_d - lambda)); //cos(theta);//cos(0.5*M_PI + atan(50./220.));
            if (cos_theta < -1)
              cos_theta = -1;
            //0.5*_PIだとnanに
            float sin_theta = sqrt(1. - pow((cos_theta), 2));

            if (abs(cos_theta) > 1.0)
            {
              //cos_theta+=0.01;
              //printf(" %f %f\n", cos_theta,sin_theta);
            }

            Energy = 511. / lambda_d;
            //mu_H2O = csv_H2O[3][(int)(Energy+0.5)]*dens_H2O;
            //mu_Ca =  csv_Ca[3][(int)(Energy+0.5)]*dens_Ca;
            mu_H2O = mua_H2O[(int)(Energy + 0.5)] * dens_H2O;
            mu_Ca = mua_Ca[(int)(Energy + 0.5)] * dens_Ca;

            //2：方位角，光路長計算
            float phi = hiprand_uniform(&st) * 2. * M_PI;
            //乗算では.を忘れるな

            //3:相対座標→絶対座標
            if (com_flag == true)
            {
              sin_theta_a = sin_theta_a_new;
              cos_theta_a = cos_theta_a_new;
              cos_phi_a = cos_phi_a_new;
            }

            cos_theta_a_new = cos_theta_a * cos_theta - sin_theta_a * sin_theta * cos(phi); //cos(0.5*M_PI) -
            if (cos_theta_a_new < -1)cos_theta_a_new = -1;
            sin_theta_a_new = sqrt(1. - pow(cos_theta_a_new, 2)); //絶対座標系の新しい角度

            cos_phi_a_new = (cos_theta_a * cos_phi_a * sin_theta * cos(phi) + sin_theta_a * cos_phi_a * cos_theta - sin_phi_a * sin_theta * sin(phi)) / sin_theta_a_new;
            sin_phi_a_new = (cos_theta_a * sin_phi_a * sin_theta * cos(phi) + sin_theta_a * sin_phi_a * cos_theta + cos_phi_a * sin_theta * sin(phi)) / sin_theta_a_new;

            if (isnan(cos_phi_a_new) || isnan(sin_phi_a_new) || isnan(cos_theta_a_new) || isnan(sin_theta_a_new))
            {
              printf("%f, %f\n", cos_theta_a, sin_theta_a);
              printf("a: %d theta: %f, %f  phi; %f , %f\n", a, cos_theta_a_new, sin_theta_a_new, cos_phi_a_new, sin_phi_a_new);
              if (com_flag)
              {
                printf("com\n");
              }
              break;
            }

            com_flag = true;

            p1.delta_sampling(mu_H2O, mu_Ca, mu_PMMA, geometry, sin_theta_a_new, cos_theta_a_new, sin_phi_a_new, cos_phi_a_new, &st);
            //delta_sampling(p, mu_H2O, mu_Ca, geometry, sin_theta_a_new, cos_theta_a_new, sin_phi_a_new, cos_phi_a_new);

            p1.theta = theta;

            float v_length = sqrt(pow(p1.length * sin_theta_a_new * cos_phi_a_new, 2) + pow(p1.length * sin_theta_a_new * sin_phi_a_new, 2) + pow(p1.length * cos_theta_a_new, 2));
            p1.before_vec0 = p1.length * sin_theta_a_new * cos_phi_a_new / v_length;
            p1.before_vec1 = p1.length * sin_theta_a_new * sin_phi_a_new / v_length;
            p1.before_vec2 = p1.length * cos_theta_a_new / v_length;
            //vectorは長さ1に正規化

            //cout<<ditector_index<<" "<<phi_a_result<<endl;

            //float x_rotate=0, y_rotate=0, x_p_rotate=0, y_p_rotate=0;
            //座標の回転は今ついてる角度と逆方向に回さねば
            /*phi_a_result = num_p;//M_PI*210./180.;
              x_rotate = p1.x*cos(-phi_a_result) - p1.y*sin(-phi_a_result);
              y_rotate = p1.x*sin(-phi_a_result) + p1.y*cos(-phi_a_result);
              x_p_rotate = p1.x_p*cos(-phi_a_result) - p1.y_p*sin(-phi_a_result);
              y_p_rotate = p1.x_p*sin(-phi_a_result) + p1.y_p*cos(-phi_a_result);*/

            //cout<<p->x<<" "<<p->y<<" "<<p->z<<endl;

            //detector_y=(10-x_p_rotate)*(y_rotate-y_p_rotate)/(x_rotate-x_p_rotate)+y_p_rotate;//x==10の時のy座標

            //float d_z=(60-p1.x_p)*(p1.z-p1.z_p)/(p1.x-p1.x_p)+p1.z_p;
            //float d_y=(60-p1.x_p)*(p1.y-p1.y_p)/(p1.x-p1.x_p)+p1.y_p;

            float x_rotate = p1.x * cosf((-num_p * M_PI) / 180.) - p1.y * sinf((-num_p * M_PI) / 180.); //phi_a_resultから変更
            float y_rotate = p1.x * sinf((-num_p * M_PI) / 180.) + p1.y * cosf((-num_p * M_PI) / 180.);
            float x_p_rotate = p1.x_p * cosf((-num_p * M_PI) / 180.) - p1.y_p * sinf((-num_p * M_PI) / 180.);
            float y_p_rotate = p1.x_p * sinf((-num_p * M_PI) / 180.) + p1.y_p * cosf((-num_p * M_PI) / 180.);

            float d_z = ((p1.z - p1.z_p) / (x_rotate - x_p_rotate)) * 60 + (x_rotate * p1.z_p - x_p_rotate * p1.z) / (x_rotate - x_p_rotate);
            float d_y = ((y_rotate - y_p_rotate) / (x_rotate - x_p_rotate)) * 60 + (x_rotate * y_p_rotate - x_p_rotate * y_rotate) / (x_rotate - x_p_rotate);

            if (x_rotate >= 60 && abs(d_z) <= 16.25 && abs(d_y) <= 16.25)
            { //&&a==q-1){// && abs(p->z)<16.25 && abs(y_rotate)<16.25){// && x_rotate>10
              //検出器を通過したかcheck,今回は必ず検出器まで到達する

              int result_y = -1 * (int(d_y * 10 - 325./2.));
              int result_x = -1 * (int(d_z * 10- 325./2.));

              //count++;

              //image[+result_y*65+result_x]++;
              atomicAdd(&image5[num_p * detector_y * detector_x + result_y * detector_y + result_x], 1);
              break;
            }
          }
        }
      }
    }
  }
  state_gpu[s_index] = st;
  
  if (err != 0)
  {
    printf("err: %d", err);
  }
  //printf(" %d ",num_add);

  /*for(int out = 0; out<325;out++){
    printf("%f\n", countp[out]);
  }*/
}

__device__ void photon::delta_sampling(float mu_H2O, float mu_Ca, float mu_PMMA,unsigned char *geometry, float sin_theta_a, float cos_theta_a, float sin_phi_a, float cos_phi_a, hiprandStateMRG32k3a *st)
{
  //printf("%f\n",x);
  float mu_max;
  //mu_max = max({mu_H2O, mu_Ca, mu_PMMA}); //多媒質の時
  mu_max = max(mu_H2O, mu_Ca);
  mu_max = max(mu_max, mu_PMMA);
  //mu_max = mu_Ca; //1媒質の場合
  //printf("%f ",mu_max);
  bool loop_flag = true;
  //bool air_flag = true;
  float x1 = x, y1 = y, z1 = z;//, length1 = length;
  float x2 = x, y2 = y, z2 = z, length3 = 0;
  x_p = x1, y_p = y1, z_p = z1;

  int num_itr = 0;
  //int geo_index;
  int check;
  if (isnan(sin_theta_a) || isnan(cos_theta_a) || isnan(sin_phi_a) || isnan(cos_phi_a))
  {
    printf("%f, %f , %f, %f\n", sin_theta_a, cos_theta_a, sin_phi_a, cos_theta_a);
    return;
  }

  while (loop_flag)
  {
    float beta = hiprand_uniform(st);
    float r = -log(beta) / mu_max;
    //printf("%f",r);

    x2 += r * sin_theta_a * cos_phi_a;
    y2 += r * sin_theta_a * sin_phi_a;
    z2 += r * cos_theta_a;
    length3 += r;
    //printf("%f, %f ,%f \n", x2,y2,z2);

    check = 0;

    //cout<<mu_Ca/mu_max;
    float nu = hiprand_uniform(st); //genrand_real3();

  //yz->xyへ
  if(x2 * x2 + y2 * y2 <= 81 && -10 <= z2 && z2 <= 10)
  {//半径9円柱(PMMA) in 3cm Ca
    //条件が多いものを上に書くことで上でif文が切れる確率が上がり，計算量短縮につながりそう
    //pow(hoge,2)からべた書きに変更．数億回計算する事を考えると気になるので変更
    /*if(int(geometry[int((z2+10)*10)*geoH*geoH + int((y2+10)*10)*geoH + int((x2+10)*10)]) == 1){
      check = 1;
    }
    else if(int(geometry[int((z2+10)*10)*geoH*geoH + int((y2+10)*10)*geoH + int((x2+10)*10)]) == 2){
        check = 2;
      }
    else if(int(geometry[int((z2+10)*10)*geoH*geoH + int((y2+10)*10)*geoH + int((x2+10)*10)]) == 0){
      check = 0;
    }
      else{
        check = 3;
      }*/
      //check = 1;
      check = int(geometry[int((z2+10)*10)*geoH*geoH + int((y2+10)*10)*geoH + int((x2+10)*10)]);
  }



    if (check == 0)
    { //空気の時
      //空気の時，直進し続ける
      if (abs(x2) >= 62 || abs(y2) >= 62 || abs(z2) >= 17)
      {
        loop_flag = false;
      }
      else
      {
        num_itr++;
      }
    }

    else if (check == 1)
    { //H2Oの時
      if (nu <= mu_H2O / mu_max)
      {
        loop_flag = false;
        num_itr++;
        //break;
      }
    }
//#if 1
    else if(check == 2){//Ca領域の時
      if(nu <=  mu_Ca/mu_max){
        loop_flag = false;
        //break;
      }
    }
//#endif
    else//PMMA
    {
      if(nu <=  mu_PMMA/mu_max){
        loop_flag = false;
        //break;
      }
    }

    if (num_itr > 100)
    {
      //printf("%d %f, %f, %f\n",num_itr, x2,y2,z2);
    }
  }

  x = x2;
  y = y2;
  z = z2;

  length = length3;
  //printf("%f\n",x);
}

__global__ void RandStateGenerator(hiprandStateMRG32k3a *state_gpu)
{
  int a = blockIdx.y * blockDim.y + threadIdx.y;
  int b = blockIdx.x * blockDim.x + threadIdx.x;


  if (a >= detector_x || b >= detector_y)
  {
    return;
  }
  int index = detector_x * a + b;

  hiprand_init(0, index, 0, &state_gpu[index]);
}

__global__ void LaunchPhoton(hiprandStateMRG32k3a *state, int seed)
{
  int a = blockIdx.y * blockDim.y + threadIdx.y;
  int b = blockIdx.x * blockDim.x + threadIdx.x;

  int index = detector_x * a + b;
  hiprand_init(seed, index, 0, &state[index]);
}

void add_result(photon *p, float phi, int *image, int *es, int count, int ditector_index, int Energy, int a, int q)
{
  float x_rotate = p->x * cos(-phi) - p->y * sin(-phi);
  float y_rotate = p->x * sin(-phi) + p->y * cos(-phi);
  float x_p_rotate = p->x_p * cos(-phi) - p->y_p * sin(-phi);
  float y_p_rotate = p->x_p * sin(-phi) + p->y_p * cos(-phi);

  float ditector_y = (10 - x_p_rotate) * (y_rotate - y_p_rotate) / (x_rotate - x_p_rotate) + y_p_rotate;

  if (x_rotate >= 16.25)
  {
    if (a != q - 1)
    { //特定の散乱回数の場合のみ検出
      return;
    }
    if (abs(ditector_y) > 10)
    {
      //cout<<ditector_index<<" "<<ditector_y<<endl;
    }
    image[ditector_index * 65 + (int)(ditector_y * 2 + 32.5)]++;
    count++;
    es[(int)Energy]++;
    return;
  }
}
